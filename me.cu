#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <math.h>
#include <stdlib.h>

#include "me.h"
#include "tables.h"

struct mv_data
{
  int sad;
  int mv_x;
  int mv_y;
};

__device__ static void sad_block_8x8(uint8_t *block1, uint8_t *block2, int stride, int *result)
{
  int u, v;

  *result = 0;

  for (v = 0; v < 8; ++v)
  {
    for (u = 0; u < 8; ++u)
    {
      *result += abs(block2[v*stride+u] - block1[v*stride+u]);
    }
  }
}

/* Motion estimation for 8x8 block */
__global__ static void me_block_8x8(struct c63_common *cm, struct macroblock *mb_gpu, uint8_t *orig, uint8_t *ref, int color_component)
{
  int mb_x = blockIdx.x;
  int mb_y = blockIdx.y;
  struct macroblock *mb = &mb_gpu[mb_y*cm->padw[color_component]/8+mb_x];

  int range = cm->me_search_range;

  /* Quarter resolution for chroma channels. */
  if (color_component > 0) { range /= 2; }

  int w = cm->padw[color_component];
  int h = cm->padh[color_component];

  int x = mb_x * 8 + threadIdx.x - range;
  int y = mb_y * 8 + threadIdx.y - range;

  // Store all SADs in a flat array such that we can find the minimun SAD later
  extern __shared__ struct mv_data sad_array[];
  
  int flattenedThreadIdx = threadIdx.y * blockDim.x + threadIdx.x;
  
  int mx = mb_x * 8;
  int my = mb_y * 8;
  int sad;

  sad_array[flattenedThreadIdx].mv_x = x - mx;
  sad_array[flattenedThreadIdx].mv_y = y - my;

  if (x < 0 || y < 0 || x > w - 8 || y > h - 8) {
    sad_array[flattenedThreadIdx].sad = INT_MAX;
  } else {
    sad_block_8x8(orig + my*w+mx, ref + y*w+x, w, &sad);

    // Store the SAD for this thread in the appropriate index in the array
    sad_array[flattenedThreadIdx].sad = sad;
  }

  __syncthreads();

  // Sequential addressing minimum algorithm
  for(int stride = (blockDim.x*blockDim.y)/2; stride > 1; stride /= 2)
  {
    // Gives out way too high values
    // Each iteration the amount of threads working will be halved since we compare 2 elements each iteration
    if(flattenedThreadIdx < stride)
    {
      if(sad_array[flattenedThreadIdx].sad > sad_array[flattenedThreadIdx + stride].sad)
      {
        sad_array[flattenedThreadIdx] = sad_array[flattenedThreadIdx + stride];
        // printf("sad_array[%d] = %d\n", flattenedThreadIdx, sad_array[flattenedThreadIdx])
      }
    }

    __syncthreads();
  }

  if (threadIdx.x == 0 && threadIdx.y == 0)
  {
    mb->mv_x = sad_array[0].mv_x;
    mb->mv_y = sad_array[0].mv_y;
    mb->use_mv = 1;
  }
}

void c63_motion_estimate(struct c63_common *cm)
{
  /* Compare this frame with previous reconstructed frame */
  struct c63_common *cm_gpu;
  struct macroblock *mb_Y, *mb_U, *mb_V;

  hipMalloc((void **)&cm_gpu, sizeof(struct c63_common));

  hipMalloc((void **)&mb_Y, sizeof(struct macroblock)*(cm->mb_rows)*(cm->mb_cols));
  hipMalloc((void **)&mb_U, sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2));
  hipMalloc((void **)&mb_V, sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2));

  hipMemcpy(cm_gpu, cm, sizeof(struct c63_common), hipMemcpyHostToDevice);

  hipMemcpy(mb_Y, cm->curframe->mbs[Y_COMPONENT], sizeof(struct macroblock)*(cm->mb_rows)*(cm->mb_cols), hipMemcpyHostToDevice);
  hipMemcpy(mb_U, cm->curframe->mbs[U_COMPONENT], sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2), hipMemcpyHostToDevice);
  hipMemcpy(mb_V, cm->curframe->mbs[V_COMPONENT], sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2), hipMemcpyHostToDevice);

  uint8_t *orig_Y, *recons_Y;
  hipMalloc((void **)&orig_Y, sizeof(uint8_t)*cm->ypw*cm->yph);
  hipMalloc((void **)&recons_Y, sizeof(uint8_t)*cm->ypw*cm->yph);

  uint8_t *orig_U, *recons_U;
  hipMalloc((void **)&orig_U, sizeof(uint8_t)*cm->upw*cm->uph);
  hipMalloc((void **)&recons_U, sizeof(uint8_t)*cm->upw*cm->uph);

  uint8_t *orig_V, *recons_V;
  hipMalloc((void **)&orig_V, sizeof(uint8_t)*cm->vpw*cm->vph);
  hipMalloc((void **)&recons_V, sizeof(uint8_t)*cm->vpw*cm->vph);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipMemcpy(orig_Y, cm->curframe->orig->Y, sizeof(uint8_t)*cm->ypw*cm->yph, hipMemcpyHostToDevice);
  hipMemcpy(orig_U, cm->curframe->orig->U, sizeof(uint8_t)*cm->upw*cm->uph, hipMemcpyHostToDevice);
  hipMemcpy(orig_V, cm->curframe->orig->V, sizeof(uint8_t)*cm->vpw*cm->vph, hipMemcpyHostToDevice);
  hipMemcpy(recons_Y, cm->curframe->recons->Y, sizeof(uint8_t)*cm->ypw*cm->yph, hipMemcpyHostToDevice);
  hipMemcpy(recons_U, cm->curframe->recons->U, sizeof(uint8_t)*cm->upw*cm->uph, hipMemcpyHostToDevice);
  hipMemcpy(recons_V, cm->curframe->recons->V, sizeof(uint8_t)*cm->vpw*cm->vph, hipMemcpyHostToDevice);

  dim3 lumaThreadsPerBlock(cm->me_search_range*2, cm->me_search_range*2);
  dim3 lumaGridDim(cm->mb_cols ,cm->mb_rows);

  /* Luma */
  me_block_8x8<<<lumaGridDim, lumaThreadsPerBlock, lumaThreadsPerBlock.x*lumaThreadsPerBlock.y*sizeof(int)*3>>>(cm_gpu, mb_Y, orig_Y, recons_Y, Y_COMPONENT);

  dim3 chromaThreadsPerBlock(cm->me_search_range, cm->me_search_range);
  dim3 chromaGridDim(cm->mb_cols/2, cm->mb_rows/2);

  /* Chroma */
  me_block_8x8<<<chromaGridDim, chromaThreadsPerBlock, chromaThreadsPerBlock.x*chromaThreadsPerBlock.y*sizeof(int)*3>>>(cm_gpu, mb_U, orig_U, recons_U, U_COMPONENT);
  me_block_8x8<<<chromaGridDim, chromaThreadsPerBlock, chromaThreadsPerBlock.x*chromaThreadsPerBlock.y*sizeof(int)*3>>>(cm_gpu, mb_V, orig_V, recons_V, V_COMPONENT);

  hipDeviceSynchronize();

  hipMemcpy(cm->curframe->mbs[Y_COMPONENT], mb_Y, sizeof(struct macroblock)*(cm->mb_rows)*(cm->mb_cols), hipMemcpyDeviceToHost);
  hipMemcpy(cm->curframe->mbs[U_COMPONENT], mb_U, sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2), hipMemcpyDeviceToHost);
  hipMemcpy(cm->curframe->mbs[V_COMPONENT], mb_V, sizeof(struct macroblock)*(cm->mb_rows/2)*(cm->mb_cols/2), hipMemcpyDeviceToHost);

  hipFree(orig_Y);
  hipFree(recons_Y);
  hipFree(orig_U);
  hipFree(recons_U);
  hipFree(orig_V);
  hipFree(recons_V);
  hipFree(cm_gpu);
  hipFree(mb_Y);
  hipFree(mb_U);
  hipFree(mb_V);
}

/* Motion compensation for 8x8 block */
static void mc_block_8x8(struct c63_common *cm, int mb_x, int mb_y,
    uint8_t *predicted, uint8_t *ref, int color_component)
{
  struct macroblock *mb =
    &cm->curframe->mbs[color_component][mb_y*cm->padw[color_component]/8+mb_x];

  if (!mb->use_mv) { return; }

  int left = mb_x * 8;
  int top = mb_y * 8;
  int right = left + 8;
  int bottom = top + 8;

  int w = cm->padw[color_component];

  /* Copy block from ref mandated by MV */
  int x, y;

  for (y = top; y < bottom; ++y)
  {
    for (x = left; x < right; ++x)
    {
      predicted[y*w+x] = ref[(y + mb->mv_y) * w + (x + mb->mv_x)];
    }
  }
}

void c63_motion_compensate(struct c63_common *cm)
{
  int mb_x, mb_y;

  /* Luma */
  for (mb_y = 0; mb_y < cm->mb_rows; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->Y,
          cm->refframe->recons->Y, Y_COMPONENT);
    }
  }

  /* Chroma */
  for (mb_y = 0; mb_y < cm->mb_rows / 2; ++mb_y)
  {
    for (mb_x = 0; mb_x < cm->mb_cols / 2; ++mb_x)
    {
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->U,
          cm->refframe->recons->U, U_COMPONENT);
      mc_block_8x8(cm, mb_x, mb_y, cm->curframe->predicted->V,
          cm->refframe->recons->V, V_COMPONENT);
    }
  }
}
